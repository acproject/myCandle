#include "hip/hip_runtime.h"
// WARNING: THIS IS ONLY VALID ASSUMING THAT inp IS CONTIGUOUS!
// TODO: proper error reporting when ids are larger than v_size.
#include "cuda_utils.cuh"
#include<stdint.h>

template<typename T, typename I>
__device__ void index_select(
    const size_t numel,
    const size_t num_dims,
    const size_t *info,
    const I *ids,
    const T *inp,
    T *out,
    const size_t left_size,
    const size_t dim_size,
    const size_t right_size
) {
    const size_t *dims = info;
    const size_t *strides = info + num_dims;
    if (is_contiguous(num_dims, dims, strides)) {
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
            for (unsigned int j = 0; j < left_size; ++j) {
                memcpy(&out[(i + j * numel) * right_size], &inp[(j * dim_size + ids[i]) * right_size], right_size * sizeof(T));
            }
        }
    }
    else {
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
            unsigned strided_i = get_strided_index(i, num_dims, dims, strides);
            for (unsigned int j = 0; j < left_size; ++j) {
                memcpy(&out[(i + j * numel) * right_size], &inp[(j * dim_size + ids[strided_i]) * right_size], right_size * sizeof(T));
            }
        }
    }
}

#define IS_OP(TYPENAME, INDEX_TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const size_t numel,  \
    const size_t num_dims, \
    const size_t *info, \
    const INDEX_TYPENAME *ids, \
    const TYPENAME *inp, \
    TYPENAME *out, \
    const size_t left_size, \
    const size_t dim_size, \
    const size_t right_size \
) { index_select(numel, num_dims, info, ids, inp, out, left_size, dim_size, right_size); } \

template<typename T, typename I>
__device__ void gather(
    const size_t numel,
    const I *ids,
    const T *inp,
    T *out,
    const size_t left_size,
    const size_t src_dim_size,
    const size_t ids_dim_size,
    const size_t right_size
) {
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
        size_t post = i % right_size;
        size_t idx = ids[i];
        size_t pre = i / (right_size * ids_dim_size);
        size_t src_i = (pre * src_dim_size + idx) * right_size + post;
        out[i] = inp[src_i];
    }
}

#define GATHER_OP(TYPENAME, INDEX_TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const size_t numel,  \
    const INDEX_TYPENAME *ids, \
    const TYPENAME *inp, \
    TYPENAME *out, \
    const size_t left_size, \
    const size_t src_dim_size, \
    const size_t ids_dim_size, \
    const size_t right_size \
) { gather(numel, ids, inp, out, left_size, src_dim_size, ids_dim_size, right_size); } \

template<typename T, typename I>
__device__ void index_add(
    const I *ids,
    const size_t ids_dim_size,
    const T *inp,
    T *out,
    const size_t left_size,
    const size_t src_dim_size,
    const size_t dst_dim_size,
    const size_t right_size
) {
      const size_t numel = left_size * right_size;
      for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
          const size_t pre = i / right_size;
          const size_t post = i % right_size;
          for (unsigned int j = 0; j < ids_dim_size; ++j) {
              const size_t idx = ids[j];
              const size_t src_i = (pre * ids_dim_size + j) * right_size + post;
              const size_t dst_i = (pre * dst_dim_size + idx) * right_size + post;
              out[dst_i] += inp[src_i];
          }
      }
}

#define IA_OP(TYPENAME, INDEX_TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const INDEX_TYPENAME *ids, \
    const size_t ids_dim_size, \
    const TYPENAME *inp, \
    TYPENAME *out, \
    const size_t left_size, \
    const size_t src_dim_size, \
    const size_t dst_dim_size, \
    const size_t right_size \
) { index_add(ids, ids_dim_size, inp, out, left_size, src_dim_size, dst_dim_size, right_size); } \

template<typename T, typename I>
__device__ void scatter_add(
    const I *ids,
    const T *inp,
    T *out,
    const size_t left_size,
    const size_t src_dim_size,
    const size_t dst_dim_size,
    const size_t right_size
) {
      const size_t numel = left_size * right_size;
      for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
          const size_t pre = i / right_size;
          const size_t post = i % right_size;
          for (unsigned int j = 0; j < src_dim_size; ++j) {
              const size_t src_i = (pre * src_dim_size + j) * right_size + post;
              const size_t idx = ids[src_i];
              const size_t dst_i = (pre * dst_dim_size + idx) * right_size + post;
              out[dst_i] += inp[src_i];
          }
      }
}

#define SA_OP(TYPENAME, INDEX_TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const INDEX_TYPENAME *ids, \
    const TYPENAME *inp, \
    TYPENAME *out, \
    const size_t left_size, \
    const size_t src_dim_size, \
    const size_t dst_dim_size, \
    const size_t right_size \
) { scatter_add(ids, inp, out, left_size, src_dim_size, dst_dim_size, right_size); } \


#if __CUDA_ARCH__ >= 800
IS_OP(__hip_bfloat16, uint32_t, is_u32_bf16)
IS_OP(__hip_bfloat16, uint8_t, is_u8_bf16)
GATHER_OP(__hip_bfloat16, uint32_t, gather_u32_bf16)
GATHER_OP(__hip_bfloat16, uint8_t, gather_u8_bf16)
IA_OP(__hip_bfloat16, uint32_t, ia_u32_bf16)
IA_OP(__hip_bfloat16, uint8_t, ia_u8_bf16)
SA_OP(__hip_bfloat16, uint32_t, sa_u32_bf16)
SA_OP(__hip_bfloat16, uint8_t, sa_u8_bf16)
#endif

#if __CUDA_ARCH__ >= 530
IS_OP(__half, uint32_t, is_u32_f16)
IS_OP(__half, uint8_t, is_u8_f16)
GATHER_OP(__half, uint32_t, gather_u32_f16)
GATHER_OP(__half, uint8_t, gather_u8_f16)
IA_OP(__half, uint32_t, ia_u32_f16)
IA_OP(__half, uint8_t, ia_u8_f16)
SA_OP(__half, uint32_t, sa_u32_f16)
SA_OP(__half, uint8_t, sa_u8_f16)
#endif

IS_OP(float, uint32_t, is_u32_f32)
IS_OP(double, uint32_t, is_u32_f64)
IS_OP(uint8_t, uint32_t, is_u32_u8)
IS_OP(uint32_t, uint32_t, is_u32_u32)

IS_OP(float, uint8_t, is_u8_f32)
IS_OP(double, uint8_t, is_u8_f64)
IS_OP(uint8_t, uint8_t, is_u8_u8)
IS_OP(uint32_t, uint8_t, is_u8_u32)

GATHER_OP(float, uint32_t, gather_u32_f32)
GATHER_OP(double, uint32_t, gather_u32_f64)
GATHER_OP(uint8_t, uint32_t, gather_u32_u8)
GATHER_OP(uint32_t, uint32_t, gather_u32_u32)

GATHER_OP(float, uint8_t, gather_u8_f32)
GATHER_OP(double, uint8_t, gather_u8_f64)
GATHER_OP(uint8_t, uint8_t, gather_u8_u8)
GATHER_OP(uint32_t, uint8_t, gather_u8_u32)

IA_OP(float, uint32_t, ia_u32_f32)
IA_OP(double, uint32_t, ia_u32_f64)
IA_OP(uint8_t, uint32_t, ia_u32_u8)
IA_OP(uint32_t, uint32_t, ia_u32_u32)

IA_OP(float, uint8_t, ia_u8_f32)
IA_OP(double, uint8_t, ia_u8_f64)
IA_OP(uint8_t, uint8_t, ia_u8_u8)
IA_OP(uint32_t, uint8_t, ia_u8_u32)

SA_OP(float, uint32_t, sa_u32_f32)
SA_OP(double, uint32_t, sa_u32_f64)
SA_OP(uint8_t, uint32_t, sa_u32_u8)
SA_OP(uint32_t, uint32_t, sa_u32_u32)

SA_OP(float, uint8_t, sa_u8_f32)
SA_OP(double, uint8_t, sa_u8_f64)
SA_OP(uint8_t, uint8_t, sa_u8_u8)
SA_OP(uint32_t, uint8_t, sa_u8_u32)
