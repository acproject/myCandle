#include "hip/hip_runtime.h"
#include "binary_op_macros.cuh"
#include<stdint.h>

#if __CUDA_ARCH__ >= 800
BINARY_OP(__hip_bfloat16, badd_bf16, x + y)
BINARY_OP(__hip_bfloat16, bdiv_bf16, x / y)
BINARY_OP(__hip_bfloat16, bmul_bf16, x * y)
BINARY_OP(__hip_bfloat16, bsub_bf16, x - y)
BINARY_OP_OUT(__hip_bfloat16, uint8_t, eq_bf16, x == y)
BINARY_OP_OUT(__hip_bfloat16, uint8_t, ne_bf16, x != y)
BINARY_OP_OUT(__hip_bfloat16, uint8_t, lt_bf16, x < y)
BINARY_OP_OUT(__hip_bfloat16, uint8_t, le_bf16, x <= y)
BINARY_OP_OUT(__hip_bfloat16, uint8_t, gt_bf16, x > y)
BINARY_OP_OUT(__hip_bfloat16, uint8_t, ge_bf16, x >= y)
#endif

#if __CUDA_ARCH__ >= 530
BINARY_OP(__half, badd_f16, x + y)
BINARY_OP(__half, bdiv_f16, x / y)
BINARY_OP(__half, bmul_f16, x * y)
BINARY_OP(__half, bsub_f16, x - y)
BINARY_OP_OUT(__half, uint8_t, eq_f16, x == y)
BINARY_OP_OUT(__half, uint8_t, ne_f16, x != y)
BINARY_OP_OUT(__half, uint8_t, lt_f16, x < y)
BINARY_OP_OUT(__half, uint8_t, le_f16, x <= y)
BINARY_OP_OUT(__half, uint8_t, gt_f16, x > y)
BINARY_OP_OUT(__half, uint8_t, ge_f16, x >= y)
#endif

BINARY_OP(float, badd_f32, x + y)
BINARY_OP(double, badd_f64, x + y);
BINARY_OP(uint8_t, badd_u8, x + y);
BINARY_OP(uint32_t, badd_u32, x + y);
BINARY_OP(float, bdiv_f32, x / y)
BINARY_OP(double, bdiv_f64, x / y);
BINARY_OP(uint8_t, bdiv_u8, x / y);
BINARY_OP(uint32_t, bdiv_u32, x / y);
BINARY_OP(float, bmul_f32, x * y)
BINARY_OP(double, bmul_f64, x * y);
BINARY_OP(uint8_t, bmul_u8, x * y);
BINARY_OP(uint32_t, bmul_u32, x * y);
BINARY_OP(float, bsub_f32, x - y)
BINARY_OP(double, bsub_f64, x - y);
BINARY_OP(uint8_t, bsub_u8, x - y);
BINARY_OP(uint32_t, bsub_u32, x - y);

BINARY_OP_OUT(float, uint8_t, eq_f32, x == y)
BINARY_OP_OUT(double, uint8_t, eq_f64, x == y)
BINARY_OP_OUT(uint8_t, uint8_t, eq_u8, x == y)
BINARY_OP_OUT(uint32_t, uint8_t, eq_u32, x == y)

BINARY_OP_OUT(float, uint8_t, ne_f32, x != y)
BINARY_OP_OUT(double, uint8_t, ne_f64, x != y)
BINARY_OP_OUT(uint8_t, uint8_t, ne_u8, x != y)
BINARY_OP_OUT(uint32_t, uint8_t, ne_u32, x != y)

BINARY_OP_OUT(float, uint8_t, lt_f32, x < y)
BINARY_OP_OUT(double, uint8_t, lt_f64, x < y)
BINARY_OP_OUT(uint8_t, uint8_t, lt_u8, x < y)
BINARY_OP_OUT(uint32_t, uint8_t, lt_u32, x < y)

BINARY_OP_OUT(float, uint8_t, le_f32, x <= y)
BINARY_OP_OUT(double, uint8_t, le_f64, x <= y)
BINARY_OP_OUT(uint8_t, uint8_t, le_u8, x <= y)
BINARY_OP_OUT(uint32_t, uint8_t, le_u32, x <= y)

BINARY_OP_OUT(float, uint8_t, gt_f32, x > y)
BINARY_OP_OUT(double, uint8_t, gt_f64, x > y)
BINARY_OP_OUT(uint8_t, uint8_t, gt_u8, x > y)
BINARY_OP_OUT(uint32_t, uint8_t, gt_u32, x > y)

BINARY_OP_OUT(float, uint8_t, ge_f32, x >= y)
BINARY_OP_OUT(double, uint8_t, ge_f64, x >= y)
BINARY_OP_OUT(uint8_t, uint8_t, ge_u8, x >= y)
BINARY_OP_OUT(uint32_t, uint8_t, ge_u32, x >= y)
