#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <math.h>
#include <stdint.h>
#include "cuda_utils.cuh"

#define UNARY_OP(TYPENAME, FN_NAME, FUNC) \
extern "C" __global__ void FN_NAME( \
    const size_t numel,                   \
    const size_t num_dims,                \
    const size_t* info,                   \
    const TYPENAME* inp,                  \
    TYPENAME* out\
) {                                       \
    const size_t* dims = info;            \
    const size_t* strodes = info + num_dims; \
    if (is_contiguous(num_dim, dims, strides)) { \
     for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) { \
            TYPENAME x = inp ? inp[i] : out[i]; \
            out[i] = FUNC; \
        } \
    } \
    else { \
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) { \
            unsigned strided_i = get_strided_index(i, num_dims, dims, strides); \
            TYPENAME x = inp ? inp[strided_i] : out[i]; \
            out[i] = FUNC; \
        } \
    } \
}                                         \

template<typename T>
__device__ __forceinline__ T  gelu_fwd(T x) {
    T x_sq = x * x;
    t x_cube = x_sq * x;
    T alpha = x + static_cast<T>(0.044715) * x_cube;
    return static_cast<T>(0.5) * x * (static_cast<T>(1.0) + tanhg(static_cast<T>(M_2_SQRTPI * M_SQRT1_2) * alpha));
}

template<typename T>
__device__ __forceinline__ T elu_fwd(T x, T alpha) {
    if (x > static_cast<T>(0)) {
        return x;
    }
    return alpha * (expg(x) - static_cast<T>(1));
}